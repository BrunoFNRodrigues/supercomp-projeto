#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/execution_policy.h>

 using namespace std::chrono;

void reportTime(const char* msg, steady_clock::duration span);

int main(int argc, char *argv[]){
     steady_clock::time_point ts, te;

    // quantidade de filmes e categorias 
    int N = 0;
    int M = 0;

    // le a primeira linha do arquivo e atribui os valores de quantidade de filmes e categorias
    std::cin >> N >> M;

    // Carregar os dados do arquivo de entrada na memória da GPU
    thrust::device_vector<int> start_times(N);
    thrust::device_vector<int> end_times(N);
    thrust::device_vector<int> categories(N);
    thrust::device_vector<int> L(M);   
    
    // varicavel para a leitura dos limetes por categorias
    int catn;

    // loop que le o limite de cada categoria e salva do vetor categorias
    for (int i = 0; i < M; i++){
        std::cin >> catn;
        L[i] = catn; 
    }

    // horario de inicio, fim e a categoria do filme
    int inicio, fim, categoria;

    // le uma linha correspondente a um filme até chegar no valor que foi passado na primeira linha do arquivo
    for (int i = 0; i < N; i++){
        std::cin >> inicio >> fim >> categoria;
        // realiza o ajuste horario caso um filme acaba em horario menor que o de inicio para ele acabar as 24
        start_times[i] = inicio;
        end_times[i] = fim;
        categories[i] = categoria;
    }

    // Criar a matriz de programação dinâmica
    thrust::device_vector<int> dp((N+1) * (M+1), 0);

    // Inicializar a primeira linha da matriz com zeros
    thrust::fill(dp.begin(), dp.begin() + M + 1, 0);
    
    ts = steady_clock::now();
    // Preencher a matriz com as soluções para subproblemas menores
    for (int i = 1; i <= N; i++) {
        for (int j = 1; j <= M; j++) {
            // Encontrar o número máximo de filmes que podem ser assistidos até o filme i e categoria j
            int max_count = 0;
            for (int k = 0; k < i; k++) {
                if (categories[k] == j && end_times[k] <= start_times[i] && dp[(k*(M+1)) + j-1] + 1 <= L[j-1]) {
                    max_count = max(max_count, dp[(k*(M+1)) + j-1] + 1);
                } else {
                    max_count = max(max_count, dp[(k*(M+1)) + j]);
                }
            }
            dp[(i*(M+1)) + j] = max_count;
        }
    }

    // Encontrar o número máximo de filmes que podem ser assistidos
    int max_count = 0;
    for (int j = 1; j <= M; j++) {
        max_count = max(max_count, dp[(N*(M+1)) + j]);
    }

    te = steady_clock::now();
    std::cout << "," << N << "," << M << "," << 0 << "," << reportTime("Tempo para calculo", te - ts) << "," << max_count << std::endl;

    // salvo os valoresde quantidade de filmes, quantidade de categorias, tempo nao alocado, tempo de execucao e quantidade de filmes alocados 
    // em um arquivo csv
    string arquivo = argv[1];
    ofstream file;
    file.open ("./../resultados/"+arquivo+".csv", ios_base::app);
    file << "," << std::to_string(N) << "," << std::to_string(M) << "," << "0" << "," << std::to_string(reportTime("Tempo para calculo", te - ts)) << "," << std::to_string(max_count) << std::endl;
    file.close();
}

void reportTime(const char* msg, steady_clock::duration span) {
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " <<
    ms.count() << " milisegundos" << std::endl;
}