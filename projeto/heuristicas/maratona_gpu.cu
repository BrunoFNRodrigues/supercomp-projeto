#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <algorithm> 
#include <random>
#include <chrono>
#include <stdlib.h> 
#include <stdio.h>
#include <iterator>
#include <random>
#include <chrono>
#include <fstream>
#include <cstdint>
#include <stack>
#include <utility>
#include <map>
#include <ctime>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>


using namespace std;

struct bestMaraton
{  
    int N;
    int M;
    int* L;
    int* start_times;
    int* end_times;
    int* categories;
    bestMaraton(int N_, int M_, int* L_, int* start_times_, int* end_times_, int* categories_) : 
    N(N_), M(M_), L(L_), start_times(start_times_), end_times(end_times_), categories(categories_) {}
    __host__ __device__
    int operator()(const int& com) {
        int L_copy[50];
        for (int i = 0; i < M; i++){
            L_copy[i] = *(L+i);
        }
        int max_count = 0;
        int time = 24;
        for (int i = 1; i < N; i++){
            if (com & (1 << i)){
                if (L_copy[i-1] > 0){
                    if(end_times[i-1] <= start_times[i]){
                        // diminui a quantidade de filmes que ainda podem ser colocadas naquela categoria
                        L_copy[categories[i-1]-1]--;
                        time -= end_times[i-1] - start_times[i-1];
                        max_count++;
                    }
                }
            }
        
        }

        return max_count;
    }
};



int main(int argc, char *argv[]){
    int N, M;
    std::cin >> N >> M;

    thrust::host_vector<int> host_categories(N);
    thrust::host_vector<int> host_start_times(N);
    thrust::host_vector<int> host_end_times(N);
    thrust::host_vector<int> host_L(M);

    for (int i = 0; i < M; i++) {
        std::cin >> host_L[i];
    }

    // horario de inicio, fim e a categoria do filme
    int start, end, categorie;

    for (int i = 0; i < N; i++) {
        std::cin >> start >> end >> categorie;
        host_start_times[i]
        host_end_times[i]
        host_categories[i]
        // realiza o ajuste horario caso um filme acaba em horario menor que o de inicio para ele acabar as 24
        if (end < start){
            end = 24;
        } 
        host_start_times[i] = start;
        host_end_times[i] = end;
        host_categories[i] = categorie;
    }

    thrust::device_vector<int> device_com(pow(2, N));
    thrust::device_vector<int> categories(host_categories);
    thrust::device_vector<int> start_times(host_start_times);
    thrust::device_vector<int> end_times(host_end_times);
    thrust::device_vector<int> L(host_L);

    thrust::sequence(device_com.begin(), device_com.end());

    // inicia a contagem do tempo de execução
    clock_t t = clock();


    thrust::transform(device_com.begin(), device_com.end(), device_com.begin(), bestMaraton(N, M, raw_pointer_cast(L.data()),
    raw_pointer_cast(start_times.data()), raw_pointer_cast(end_times.data()), raw_pointer_cast(categories.data())));
    

    
    thrust::host_vector<int> host_com = device_com;
    
    int max_count = 0;
    int iters = pow(2, N);
    for (int i = 0; i < iters; i++){
        if (host_com[i] > max_count){
            max_count = host_com[i];
        }
    }

    // termina de contar o tempo de execucao
    t = clock() - t;   
    // salvo os valoresde quantidade de filmes, quantidade de categorias, tempo nao alocado, tempo de execucao e quantidade de filmes alocados 
    // em um arquivo csv
    string arquivo = argv[1];
    ofstream file;
    file.open ("./../resultados/"+arquivo+".csv", ios_base::app);
    file << to_string(N)+","+to_string(M)+","+to_string(0)+","+to_string(((float)t)/CLOCKS_PER_SEC)+","+to_string(max_count) << endl;
    file.close();
}